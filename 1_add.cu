// Add with a single thread on the GPU


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c)  {
  *c = a + b;
}

int main() {
  int c;          // host copies
  int *dev_c;  // device copies
  int size = sizeof(int);

  // Allocate space on device
  hipMalloc((void **) &dev_c, size);


  // Launch add() on GPU
  add<<<1,1>>>(8, 2, dev_c);

  // Copy result to host
  hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

  printf("%d\n", c);

  // Cleanup
  hipFree(dev_c);

  return 0;
}
